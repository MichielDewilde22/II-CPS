#include "hip/hip_runtime.h"

/*
 * Delay and sum beamforming for eRTIS using CUDA accelerated kernels.
 *
 * arguments: Delay matrix [( directions * channels),type int32] and matched filter matrix [( samples * channels), type single].
 * 
 * Compile with 'mexcuda -v mex_fast_SpatioTemp_MF_CUDA.cu' (-v for extra details for debugging)
 * Requires CUDA toolkit (9.1 was used by me) and visual studio (2015 was used by me).
 * Make sure to correctly set c++ compiler with 'mex -setup c++' and clicking on the link of the version you want.
 * And make sure to set the CUDA enviroment variable correctly with
 * 'setenv('MW_NVCC_PATH','C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\vX\bin')'
 *
 * Wouter Jansen
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <string>     
        
/*
 * Device code for GPU kernel to calculate beamforming. 
 * With threads going over all 3 dimensions: directions, samples and microphones.
 */
void __global__ beamform_kernel(int const *delay_matrix, float const *dataMatchedFilter,float *dataBeamform,
     int nmicrophones, int ndirections, int nspsls, int output_size, int sample_size){
    int output_size_block = blockIdx.x * blockDim.x + threadIdx.x;
    int direction = blockIdx.y * blockDim.y + threadIdx.y;
    int microphone = blockIdx.z * blockDim.z + threadIdx.z;
    if(direction < ndirections && output_size_block < sample_size && microphone < nmicrophones){
            atomicAdd(&dataBeamform[output_size_block + direction*output_size], dataMatchedFilter[microphone*nspsls + delay_matrix[microphone*ndirections + direction] + output_size_block]);
    }
}

/*
 * Host code for CPU
 */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray *prhs[])
{
    /* Declare all variables.*/
    mxGPUArray const *dataMatchedFilter;
    mxGPUArray const *delayMatrix;
    int * delayMatrixCPU;
    mxGPUArray *dataBeamForm;
    float const *d_dataMatchedFilter;
    int const *d_delayMatrix;
    float *d_dataBeamForm;
    int nmicrophones;
    int ndirections;
    int nspsls;
    int outputSize;
    int sampleSize;
    int maxDelay = 0;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input are not a CPU arrays. */
    if ( (mxIsGPUArray(prhs[0])) || (mxIsGPUArray(prhs[1]))) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "The input matrices have to be normal CPU arrays, not GPUArrays.\n");
    }

    /* Throw an error if the input are not the correct datatype. */
    if ( mxGetClassID(prhs[0]) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "The matched filter data matrix has to be of datatype 'single'.\n");
    }
    if ( mxGetClassID (prhs[1]) !=  mxINT32_CLASS) {
        mexErrMsgIdAndTxt("parallel:gpu:mexGPUExample:InvalidInput", "The delay matrix has to be of datatype 'int32'.\n");
    }

    dataMatchedFilter = mxGPUCreateFromMxArray(prhs[0]);
    delayMatrix = mxGPUCreateFromMxArray(prhs[1]);
    delayMatrixCPU = (int *)mxGetData( prhs[1] );

    nmicrophones = mxGPUGetDimensions(dataMatchedFilter)[1];
    nspsls = mxGPUGetDimensions(dataMatchedFilter)[0];
    ndirections = mxGPUGetDimensions(delayMatrix)[0];

    /* Extract a pointer to the input data on the device. */
    d_dataMatchedFilter = (float const *)(mxGPUGetDataReadOnly(dataMatchedFilter));
    d_delayMatrix = (int const *)(mxGPUGetDataReadOnly(delayMatrix));

    /* Calculate the maximum delay and set the output size. */
    for( int cnt = 0; cnt < nmicrophones * ndirections; cnt ++ ){
        if( delayMatrixCPU[ cnt ] > maxDelay ){
            maxDelay = delayMatrixCPU[ cnt ];
        }
    }
    outputSize = nspsls + maxDelay * 4;
    sampleSize = nspsls - 2 * maxDelay;

//     printf("microphones:%i directions:%i samples:%i output size:%i sample size:%i\n",nmicrophones, ndirections, nspsls, outputSize, sampleSize);

    /* Create a GPUArray to hold the result and get its underlying pointer. */
    mwSize dims[2] = {outputSize,ndirections };
    dataBeamForm = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(dataMatchedFilter),
                            dims ,
                            mxGPUGetClassID(dataMatchedFilter),
                            mxGPUGetComplexity(dataMatchedFilter),
                            MX_GPU_INITIALIZE_VALUES );
    d_dataBeamForm = (float *)(mxGPUGetData(dataBeamForm));

    /* Execute the beamform kernel. */
    dim3 threadsPerBlock(32, 8, 4);
    dim3 numBlocks(ceil(sampleSize / (threadsPerBlock.x*1.0)), ceil(ndirections / (threadsPerBlock.y*1.0)), ceil(nmicrophones / (threadsPerBlock.z*1.0)));
    beamform_kernel<<<numBlocks, threadsPerBlock>>>(d_delayMatrix, d_dataMatchedFilter, d_dataBeamForm , nmicrophones, ndirections, nspsls, outputSize, sampleSize);

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(dataBeamForm);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(dataMatchedFilter);
    mxGPUDestroyGPUArray(delayMatrix);
    mxGPUDestroyGPUArray(dataBeamForm);
}
